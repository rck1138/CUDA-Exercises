#include "hip/hip_runtime.h"
/* histogram.cu                                
 * Kernel routine to create a histogram from an
 * array of input data.  Within a given block each
 * thread creates a private histogram.  The histograms
 * are then reduced across threads to give one histogram
 * per block.  The block-level histograms are packed 
 * into an array an returned to the CPU for the final 
 * reduction step.
 * Input:  data
 * Output: histo
 */
#include "dims.h"

__global__ void histogram(float *data, int *histo)
{
	// insert kernel body here
}
			
		
