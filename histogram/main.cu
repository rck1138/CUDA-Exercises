/* main.cu 
 * ----------------------------------------------------
 * Main routine for the dot-product example from the 
 * summer 2011 Intro to CUDA course taught at NCAR.
 * Author: Rory Kelly (rory@ucar.edu)
 * Created: 8 March 2011
 * ----------------------------------------------------
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "../common/util.h"
#include "dims.h"
#include "histogram.h"

/* local function declarations */

int main(int argc, char **argv)
{

	/* data on the CPU */
	float *h_vec1;      // input vector 
	float *h_vec2;      // input vector
        float *h_part;      // partial result
	float dp;           // dot product

	/* arrays to hold the vectors on GPU */
	float *d_vec1;      // input vector
	float *d_vec2;      // input vector
	float *d_part;      // partial result

	/* allocate CPU memory */
	h_vec1 = (float *) malloc(len*sizeof(float));
	h_vec2 = (float *) malloc(len*sizeof(float));
	h_part = (float *) malloc(blocksPerGrid*sizeof(float));

	/* allocate GPU memory */
	hipMalloc((void **) &d_vec1, len*sizeof(float));
	hipMalloc((void **) &d_vec2, len*sizeof(float));
	hipMalloc((void **) &d_part, blocksPerGrid*sizeof(float));

	/* local vars */
	int idx;

        /* get some basic info about available devices */
        printDevInfo();
	
	/* initialize vectors on CPU */
        /* dot product should sum to */
        /* len / 2.0                 */
	for(idx=0; idx<len; idx++){
		h_vec1[idx] = 0.5f;
		h_vec2[idx] = 1.0f;
	}

        /* copy memory to device array */
	hipMemcpy(d_vec1, h_vec1, len*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vec2, h_vec2, len*sizeof(float), hipMemcpyHostToDevice);

	/* call kernel */
	dot_prod<<<blocksPerGrid, threadsPerBlock>>>(d_vec1, d_vec2, d_part);

	/* copy data back to host */
	hipMemcpy(h_part, d_part, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

	/* complete sum on CPU */
	dp=0.0;
	for(idx=0; idx<blocksPerGrid; idx++)
		dp+=h_part[idx];
		
	/* print results */
	printf("----------------------------------------------\n");
	printf("Vector length: %d\n", len);
	printf("Expected value:   %8.2f\n", ((float)len)/2.0);
	printf("Calcualted value: %8.2f\n", dp);
        printf("----------------------------------------------\n");

	
        /* clean up memory on host and device */
	hipFree(d_vec1);
	hipFree(d_vec2);
	hipFree(d_part);
	free(h_vec1);
	free(h_vec2);
	free(h_part);

	return(0);
}
