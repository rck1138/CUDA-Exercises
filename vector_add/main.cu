#include "hip/hip_runtime.h"
/* hello.cu
 * A Hello World example in CUDA
 * ----------------------------------------------------
 * Example 1 from the summer 2011 Intro to CUDA course 
 * taught at NCAR.  
 * Author: Rory Kelly (rory@ucar.edu)
 * Created: 8 March 2011
 * ----------------------------------------------------
 * Example program showing the summation of two vectors
 * on the GPU.
 * ----------------------------------------------------
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "../common/util.h"
#include "vec_add.h"

/* local function declarations */
void vec_print(float *v, int len);
void vector_add(float *v1, float *v2, float *v3, int len);
float max_diff(float *v1, float *v2, int len);

int main(int argc, char **argv)
{

	/* vector length */
	int len=18000;

	/* data on the CPU to be added */
	float *h_vec1;
	float *h_vec2;
	float *h_vec3;
	float *result;

	/* arrays to hold the vectors on GPU */
	float *d_vec1;
	float *d_vec2;
	float *d_vec3;

	/* allocate CPU memory */
	h_vec1 = (float *) malloc(len*sizeof(float));
	h_vec2 = (float *) malloc(len*sizeof(float));
	h_vec3 = (float *) malloc(len*sizeof(float));
	result = (float *) malloc(len*sizeof(float));

	/* allocate GPU memory */
	hipMalloc((void **) &d_vec1, len*sizeof(float));
	hipMalloc((void **) &d_vec2, len*sizeof(float));
	hipMalloc((void **) &d_vec3, len*sizeof(float));

	/* local vars */
	int idx;

        /* get some basic info about available devices */
        printDevInfo();
	
	/* initialize vectors on CPU */
	for(idx=0; idx<len; idx++){
		h_vec1[idx] = (float)idx;
		h_vec2[idx] = (float)(-1 * idx + 1);
	}

	/* perform sum on CPU for validation */
	vector_add(h_vec1, h_vec2, result, len);

        /* copy memory to device array */
	hipMemcpy(d_vec1, h_vec1, len*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_vec2, h_vec2, len*sizeof(float), hipMemcpyHostToDevice);

	/* call kernel */
	vec_add<<<16,64>>>(d_vec1, d_vec2, d_vec3, len);

	/* copy data back to host */
	hipMemcpy(h_vec3, d_vec3, len*sizeof(float), hipMemcpyDeviceToHost);

	/* print contents of arrays */
	if(len <= 20){
		vec_print(result, len);
		vec_print(h_vec3, len);
	}
	printf("----------------------------------------------\n");
	printf("Difference between CPU and GPU Results: %6.4f\n", max_diff(result, h_vec3, len));
        printf("----------------------------------------------\n");

	
        /* clean up memory on host and device */
	//hipFree(d_data);
	free(h_vec1);
	free(h_vec2);
	free(h_vec3);

	return(0);
}

/* Routine to add 2 vectors of length len and return the 
 * result in a third vector.
 * Input vectors: v1, v2
 * Output vector: v3
 */
void vector_add(float *v1, float *v2, float *v3, int len)
{
	int i;
	for (i=0; i<len; i++)
		v3[i] = v1[i] + v2[i];
	return;
}

/* routine to print contents of vector */
void vec_print(float *v, int len)
{
	int i;
	for(i=0; i<len; i++)
		printf("%6.2f ", v[i]);
	printf("\n");
}

/* routine to find the maximum difference  */
/* between two vectors                     */
float max_diff(float *v1, float *v2, int len)
{
	int i;
	float abdiff;
	float maxd = 0.0f;

	for(i=0; i<len; i++){
		abdiff = abs(v1[i] - v2[i]);
		if(abdiff > maxd) maxd = abdiff;
	}
	return maxd;
}
