#include "hip/hip_runtime.h"
/*
 * Kernel routine to add to matrices
 */
#include "dims.h"
__global__ void matrix_add(float *m1, float *m2, float *mout)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid < size*size){
		mout[tid] = m1[tid]+m2[tid];
		tid += blockDim.x;
	}
}

__global__ void matrix_add_2d(float *m1, float *m2, float *mout)
{	
	//insert kernel body here
}
